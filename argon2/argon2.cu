#include "hip/hip_runtime.h"

/*
extern "C" {

}
*/
#include "argon2ref/argon2.h"
#include "merkletree\mtp.h"

//#include "merkletree\sha.h"
//#include "merkletree\merkletree.h"
#include "miner.h"
#include "cuda_helper.h"
#define memcost 2*1024*1024
static uint32_t* d_hash[MAX_GPUS];


extern void argon2_cpu_init(int thr_id, uint32_t threads);

extern uint32_t argon2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void argon2_setBlockTarget(const void* pDataIn, const void *pTargetIn, const void * zElement);
extern void mtp_fill(const uint64_t *Block, const uint64_t zblockHistory, uint32_t offset);
#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 8*sizeof(uint32_t)); \
		hipMemcpy(debugbuf, d_hash[thr_id], 8*sizeof(uint32_t), hipMemcpyDeviceToHost); \
		printf("lyra %s %08x %08x %08x %08x...\n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif
#define HASHLEN 32
#define SALTLEN 16
#define PWD "password"


static bool init[MAX_GPUS] = { 0 };
static __thread uint32_t throughput = 0;

extern "C" int scanhash_argon2(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done, struct mtp* mtp)
{
//	argon2_context context;
//	argon2_instance_t instance;
	uint256 TheMerkleRoot;
    
//	mtp = (struct mtp*)malloc(140*2471*sizeof(unsigned char*));

	merkletree *TheTree = new merkletree;

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		ptarget[7] = 0x00ff;

		uint32_t diff = 5;
		//((uint32_t*)pdata)[19] = 0; //start fresh
		uint32_t TheNonce;// = ((uint32_t*)pdata)[19];


	if (!init[thr_id])
	{
		int dev_id = device_map[thr_id];
		hipSetDevice(dev_id);
		
//		CUDA_LOG_ERROR();
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

		int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 17 : 16;
		if (device_sm[device_map[thr_id]] == 500) intensity = 15;
		intensity = 1;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
		throughput =  1024*64;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);


		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

printf("coming here 1 \n");
		argon2_cpu_init(thr_id, throughput);
printf("coming here 2 \n");
		init[thr_id] = true;
	}

	uint32_t _ALIGN(128) endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);
/*
	ptarget[6] = 0x0ffffff;
	ptarget[7] = 0x0ffffff;
*/	

//	argon2_initTree_cpu(thr_id, 1, endiandata[19]);

//	uint32_t diff = 5;
	((uint32_t*)pdata)[19]=0; //start fresh
	TheNonce = ((uint32_t*)pdata)[19];
//	unsigned char  TheMerkleRoot[32] = { 0 };
	argon2_context context = init_argon2d_param((const char*)pdata);
	argon2_instance_t instance;
	argon2_ctx_from_mtp(&context, &instance);
	TheTree[0] = mtp_init_withtree(TheNonce, &instance, diff, TheMerkleRoot);
	printf("fill Gpu memory\n");	

	argon2_setBlockTarget(pdata,ptarget,&TheMerkleRoot);
for (int i=0;i<memcost;i++)
	mtp_fill(instance.memory[i].v,instance.memory[i].prev_block,i);

 do   {
		int order = 0;
		uint32_t foundNonce;



		*hashes_done = pdata[19] - first_nonce + throughput;
	  
		foundNonce = argon2_cpu_hash_32(thr_id, throughput, pdata[19], d_hash[thr_id]); 
//		foundNonce = endiandata[19];
//		foundNonce = pdata[19];
		
		if (foundNonce != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash64[8];
			block_mtpProof TheBlocksAndProofs[140];
			uint256 TheUint256Target[1];
			TheUint256Target[0] = ((uint256*)ptarget)[0];
//			be32enc(&endiandata[19], foundNonce); 
//			endiandata[19] = foundNonce;
//			argon2_hash_v2(vhash64, endiandata);  
//			uint32_t is_sol = mtp_solver(foundNonce, &instance, diff,(char*)vhash64, TheMerkleRoot);
			uint32_t is_sol = mtp_solver_withblock(foundNonce, &instance, diff, TheBlocksAndProofs, TheMerkleRoot,TheTree[0],TheUint256Target[0]);
//			vhash64[7]=0;

//			if (vhash64[7] <= ptarget[7] /*&& fulltest(vhash64, ptarget)*/) {
			if (is_sol==1 /*&& fulltest(vhash64, ptarget)*/) {
				int res = 1;
				work_set_target_ratio(work, vhash64);		
				be32enc(&pdata[19], foundNonce);
				pdata[19] = foundNonce;
				for (uint32_t i=0;i<mtp_block_num;i++)
					for (uint32_t j=0;j<mtp_block_size;j++)
								((uchar*)mtp->mtpproof[i])[j] = ((uchar*)TheBlocksAndProofs)[mtp_block_size*i+j];
printf("last block \n");
for (int i=0;i<2471;i++)
   printf(" %02x ",((uchar*)mtp->mtpproof[80])[i]); 

printf("end last block \n");
				free_memory(&context, (unsigned char *)instance.memory, instance.memory_blocks, sizeof(block));
				delete TheTree;
				return res;

			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}
/*
		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
*/
		pdata[19] += throughput;
		be32enc(&endiandata[19], pdata[19]);
	}   while (!work_restart[thr_id].restart && pdata[19]<0xeffffff);
	free_memory(&context, (unsigned char *)instance.memory, instance.memory_blocks, sizeof(block));
	*hashes_done = pdata[19] - first_nonce;
	delete TheTree;
	return 0;
}


