#include "hip/hip_runtime.h"
/**
 * Argon2 based on 
 * djm34 2017
 **/

#include <stdio.h>
#include <memory.h>

#include "cuda_sha256_helper.cuh" // sha256 transform

static uint32_t *h_GNonces[16]; // this need to get fixed as the rest of that routine
static uint32_t *d_GNonces[16];

static uint32_t *h_MinNonces[16]; // this need to get fixed as the rest of that routine
static uint32_t *d_MinNonces[16];

__constant__ uint32_t pTarget[8];
__constant__ uint32_t pData[8]; // truncated data
__constant__ uint2 initMess[25];
__constant__ uint4 Elements[2];
uint4 * HBlock;
uint4 * HSmallBlock;
uint2 * HBlockHistory;
//uint4 * Elements;
uint4 * YElements;


#define ARGON2_SYNC_POINTS 4
#define argon_outlen 32
#define argon_timecost 1
#define argon_memcost 2*1024*1024 //32*1024*2 //1024*256*1 //2Gb
#define argon_lanes 4
#define argon_threads 1
#define argon_hashlen 80
#define argon_version 19
#define argon_type 0 // argon2d
#define argon_pwdlen 80 // hash and salt lenght
#define argon_default_flags 0 // hmm not sure
#define argon_segment_length argon_memcost/(argon_lanes * ARGON2_SYNC_POINTS)
#define argon_lane_length argon_segment_length * ARGON2_SYNC_POINTS
#define TREE_LEVELS 20
#define ELEM_MAX 2048
#define gpu_thread 2
#define gpu_shared 128
#define kernel1_thread 64
#define mtp_L 70
#define TPB52 32
#define TPB30 160
#define TPB20 160

__constant__ const uint2 blakeInit[8] =  
{
	{ 0xf2bdc948UL, 0x6a09e667UL },
	{ 0x84caa73bUL, 0xbb67ae85UL },
	{ 0xfe94f82bUL, 0x3c6ef372UL },
	{ 0x5f1d36f1UL, 0xa54ff53aUL },
	{ 0xade682d1UL, 0x510e527fUL },
	{ 0x2b3e6c1fUL, 0x9b05688cUL },
	{ 0xfb41bd6bUL, 0x1f83d9abUL },
	{ 0x137e2179UL, 0x5be0cd19UL }
};

__constant__ const uint2 blakeFinal[8] =  
{
	{ 0xf2bdc928UL, 0x6a09e667UL },
	{ 0x84caa73bUL, 0xbb67ae85UL },
	{ 0xfe94f82bUL, 0x3c6ef372UL },
	{ 0x5f1d36f1UL, 0xa54ff53aUL },
	{ 0xade682d1UL, 0x510e527fUL },
	{ 0x2b3e6c1fUL, 0x9b05688cUL },
	{ 0xfb41bd6bUL, 0x1f83d9abUL },
	{ 0x137e2179UL, 0x5be0cd19UL }
};

__constant__ const uint2 blakeIV[8] =
{
	{ 0xf3bcc908UL, 0x6a09e667UL },
	{ 0x84caa73bUL, 0xbb67ae85UL },
	{ 0xfe94f82bUL, 0x3c6ef372UL },
	{ 0x5f1d36f1UL, 0xa54ff53aUL },
	{ 0xade682d1UL, 0x510e527fUL },
	{ 0x2b3e6c1fUL, 0x9b05688cUL },
	{ 0xfb41bd6bUL, 0x1f83d9abUL },
	{ 0x137e2179UL, 0x5be0cd19UL }
};


__constant__ static const uint8_t blake2b_sigma[12][16] =
{
	{ 0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 } ,
	{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 } ,
	{ 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 } ,
	{ 7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 } ,
	{ 9,  0,  5,  7,  2,  4, 10, 15, 14,  1, 11, 12,  6,  8,  3, 13 } ,
	{ 2, 12,  6, 10,  0, 11,  8,  3,  4, 13,  7,  5, 15, 14,  1,  9 } ,
	{ 12,  5,  1, 15, 14, 13,  4, 10,  0,  7,  6,  3,  9,  2,  8, 11 } ,
	{ 13, 11,  7, 14, 12,  1,  3,  9,  5,  0, 15,  4,  8,  6,  2, 10 } ,
	{ 6, 15, 14,  9, 11,  3,  0,  8, 12,  2, 13,  7,  1,  4, 10,  5 } ,
	{ 10,  2,  8,  4,  7,  6,  1,  5, 15, 11,  9, 14,  3, 12, 13 , 0 } ,
	{ 0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 } ,
	{ 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 }
};

static __device__ __forceinline__ uint2 eorswap32(uint2 u, uint2 v) {
	uint2 result;
	result.y = u.x ^ v.x;
	result.x = u.y ^ v.y;
	return result;
}

__device__ static uint2 fBlaMka(uint2 x, uint2 y)
{ // hmm
uint64_t ret = devectorize(x+y);
uint64_t xy = (uint64_t)x.x * (uint64_t)y.x;
xy <<= 1;
return (vectorize(ret+xy));
}

__device__ static void fill_block4_doubleshared_mtp_output(uint4  * output,const uint4  * __restrict__ block /*, uint32_t *blockHistory*/, uint32_t prev_block_offset, uint32_t ref_block_offset)
{
	//	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	__shared__ uint2 blockR[128];
	__shared__ uint2 block_tmp[128];
	uint32_t shift2 = 128 * (threadIdx.x / 32);
	uint32_t shift = 64 * (threadIdx.x / 32);
	uint32_t itu4 = 64 / 32; //(gpu_shared/8);
	uint32_t itu2 = 128 / 32; //(gpu_shared/8);
							  //	if (threadIdx.x < 32) return;

	for (int i = 0; i<itu4; i++)
		((uint4*)blockR)[shift + itu4 * (threadIdx.x % 32) + i] = block[itu4 * (threadIdx.x % 32) + i + 64 * ref_block_offset];

	for (int i = 0; i<itu4; i++)
		((uint4*)blockR)[shift + itu4 * (threadIdx.x % 32) + i] ^= block[itu4 * (threadIdx.x % 32) + i + 64 * prev_block_offset];

		for (int i = 0; i<itu4; i++)
			((uint4*)block_tmp)[shift + itu4 * (threadIdx.x % 32) + i] = ((uint4*)blockR)[shift + itu4 * (threadIdx.x % 32) + i];

	__syncthreads();

#define G(a,b,c,d) \
   { \
     a = fBlaMka(a,b); \
     d = eorswap32(d ,a); \
     c = fBlaMka(c,d); \
     b = ROR2(b ^ c, 24); \
     a = fBlaMka(a,b); \
     d = ROR16(d ^ a); \
     c = fBlaMka(c,d); \
     b = ROR2(b ^ c, 63); \
  } 

#define BLAKE2_ROUND_NOMSG(v0, v1, v2, v3, v4, v5, v6, v7, v8, v9, v10, v11,   \
                           v12, v13, v14, v15)  \
  { \
    G(v0,v4,v8,v12); \
    G(v1,v5,v9,v13); \
    G(v2,v6,v10,v14); \
    G(v3,v7,v11,v15); \
    G(v0,v5,v10,v15); \
    G(v1,v6,v11,v12); \
    G(v2,v7,v8,v13); \
    G(v3,v4,v9,v14); \
  }

	/* Apply Blake2 on columns of 64-bit words: (0,1,...,15) , then
	(16,17,..31)... finally (112,113,...127) */
	if ((threadIdx.x % 32) <8) {

		//		for (int i = 0; i < 8; i++) {
		{
			int i = threadIdx.x % 32;
			BLAKE2_ROUND_NOMSG(
				blockR[shift2 + 16 * i], blockR[shift2 + 16 * i + 1], blockR[shift2 + 16 * i + 2],
				blockR[shift2 + 16 * i + 3], blockR[shift2 + 16 * i + 4], blockR[shift2 + 16 * i + 5],
				blockR[shift2 + 16 * i + 6], blockR[shift2 + 16 * i + 7], blockR[shift2 + 16 * i + 8],
				blockR[shift2 + 16 * i + 9], blockR[shift2 + 16 * i + 10], blockR[shift2 + 16 * i + 11],
				blockR[shift2 + 16 * i + 12], blockR[shift2 + 16 * i + 13], blockR[shift2 + 16 * i + 14],
				blockR[shift2 + 16 * i + 15]);
		}

		/* Apply Blake2 on rows of 64-bit words: (0,1,16,17,...112,113), then
		(2,3,18,19,...,114,115).. finally (14,15,30,31,...,126,127) */


		__syncthreads();
		{
			//	for (int i = 0; i < 8; i++) {
			int i = threadIdx.x % 32;
			BLAKE2_ROUND_NOMSG(
				blockR[shift2 + 2 * i], blockR[shift2 + 2 * i + 1], blockR[shift2 + 2 * i + 16],
				blockR[shift2 + 2 * i + 17], blockR[shift2 + 2 * i + 32], blockR[shift2 + 2 * i + 33],
				blockR[shift2 + 2 * i + 48], blockR[shift2 + 2 * i + 49], blockR[shift2 + 2 * i + 64],
				blockR[shift2 + 2 * i + 65], blockR[shift2 + 2 * i + 80], blockR[shift2 + 2 * i + 81],
				blockR[shift2 + 2 * i + 96], blockR[shift2 + 2 * i + 97], blockR[shift2 + 2 * i + 112],
				blockR[shift2 + 2 * i + 113]);
		}
	}

	__syncthreads();
	for (int i = 0; i<itu2; i++)
		block_tmp[shift2 + itu2 * (threadIdx.x % 32) + i] ^= blockR[shift2 + itu2 * (threadIdx.x % 32) + i];



	for (int i = 0; i<itu4; i++)
		output[itu4 * (threadIdx.x % 32) + i] = ((uint4*)block_tmp)[shift + itu4 * (threadIdx.x % 32) + i];
 
	__syncthreads();
#undef G 
#undef BLAKE2_ROUND_NOMSG
}

__device__ static void fill_block4_doubleshared_mtp_output_bhdr(uint4  * output, const uint4  * __restrict__ block /*, uint32_t *blockHistory*/, 
uint32_t prev_block_offset, uint32_t ref_block_offset)
{
		uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
		uint32 pDat[4]={0};
	__shared__ uint2 blockR[128];
	__shared__ uint2 block_tmp[128];
	uint32_t shift2 = 128 * (threadIdx.x / 32);
	uint32_t shift = 64 * (threadIdx.x / 32);
	uint32_t itu4 = 64 / 32; //(gpu_shared/8);
	uint32_t itu2 = 128 / 32; //(gpu_shared/8);
							  //	if (threadIdx.x < 32) return;

	for (int i = 0; i<itu4; i++)
		((uint4*)blockR)[shift + itu4 * (threadIdx.x % 32) + i] = block[itu4 * (threadIdx.x % 32) + i + 64 * ref_block_offset];

	for (int i = 0; i<itu4; i++)
		((uint4*)blockR)[shift + itu4 * (threadIdx.x % 32) + i] ^= block[itu4 * (threadIdx.x % 32) + i + 64 * prev_block_offset];

	for (int i = 0; i<itu4; i++)
		((uint4*)block_tmp)[shift + itu4 * (threadIdx.x % 32) + i] = ((uint4*)blockR)[shift + itu4 * (threadIdx.x % 32) + i];

	__syncthreads();


	for (int i = 0; i<itu4; i++)
	if ((shift + itu4 * (threadIdx.x % 32) + i)==8)
		((uint4*)blockR)[shift + itu4 * (threadIdx.x % 32) + i] = ((uint4*)pData)[0];
//if (thread==0)
//		printf("GPU pdat %08x %08x %08x %08x\n",pData[0],pData[1],pData[2],pData[3]);

//
	__syncthreads();

#define G(a,b,c,d) \
   { \
     a = fBlaMka(a,b); \
     d = eorswap32(d ,a); \
     c = fBlaMka(c,d); \
     b = ROR2(b ^ c, 24); \
     a = fBlaMka(a,b); \
     d = ROR16(d ^ a); \
     c = fBlaMka(c,d); \
     b = ROR2(b ^ c, 63); \
  } 

#define BLAKE2_ROUND_NOMSG(v0, v1, v2, v3, v4, v5, v6, v7, v8, v9, v10, v11,   \
                           v12, v13, v14, v15)  \
  { \
    G(v0,v4,v8,v12); \
    G(v1,v5,v9,v13); \
    G(v2,v6,v10,v14); \
    G(v3,v7,v11,v15); \
    G(v0,v5,v10,v15); \
    G(v1,v6,v11,v12); \
    G(v2,v7,v8,v13); \
    G(v3,v4,v9,v14); \
  }

	/* Apply Blake2 on columns of 64-bit words: (0,1,...,15) , then
	(16,17,..31)... finally (112,113,...127) */
	if ((threadIdx.x % 32) <8) {

		//		for (int i = 0; i < 8; i++) {
		{
			int i = threadIdx.x % 32;
			BLAKE2_ROUND_NOMSG(
				blockR[shift2 + 16 * i], blockR[shift2 + 16 * i + 1], blockR[shift2 + 16 * i + 2],
				blockR[shift2 + 16 * i + 3], blockR[shift2 + 16 * i + 4], blockR[shift2 + 16 * i + 5],
				blockR[shift2 + 16 * i + 6], blockR[shift2 + 16 * i + 7], blockR[shift2 + 16 * i + 8],
				blockR[shift2 + 16 * i + 9], blockR[shift2 + 16 * i + 10], blockR[shift2 + 16 * i + 11],
				blockR[shift2 + 16 * i + 12], blockR[shift2 + 16 * i + 13], blockR[shift2 + 16 * i + 14],
				blockR[shift2 + 16 * i + 15]);
		}

		/* Apply Blake2 on rows of 64-bit words: (0,1,16,17,...112,113), then
		(2,3,18,19,...,114,115).. finally (14,15,30,31,...,126,127) */


		__syncthreads();
		{
			//	for (int i = 0; i < 8; i++) {
			int i = threadIdx.x % 32;
			BLAKE2_ROUND_NOMSG(
				blockR[shift2 + 2 * i], blockR[shift2 + 2 * i + 1], blockR[shift2 + 2 * i + 16],
				blockR[shift2 + 2 * i + 17], blockR[shift2 + 2 * i + 32], blockR[shift2 + 2 * i + 33],
				blockR[shift2 + 2 * i + 48], blockR[shift2 + 2 * i + 49], blockR[shift2 + 2 * i + 64],
				blockR[shift2 + 2 * i + 65], blockR[shift2 + 2 * i + 80], blockR[shift2 + 2 * i + 81],
				blockR[shift2 + 2 * i + 96], blockR[shift2 + 2 * i + 97], blockR[shift2 + 2 * i + 112],
				blockR[shift2 + 2 * i + 113]);
		}
	}

	__syncthreads();
	for (int i = 0; i<itu2; i++)
		block_tmp[shift2 + itu2 * (threadIdx.x % 32) + i] ^= blockR[shift2 + itu2 * (threadIdx.x % 32) + i];



	for (int i = 0; i<itu4; i++)
		output[itu4 * (threadIdx.x % 32) + i] = ((uint4*)block_tmp)[shift + itu4 * (threadIdx.x % 32) + i];

	__syncthreads();
#undef G 
#undef BLAKE2_ROUND_NOMSG
}



__global__
void mtp_yloop(uint32_t threads, uint32_t startNounce, const uint4  * __restrict__ DBlock, const uint2  * __restrict__ DBlockHistory,
 const uint4 * __restrict__ MerkleRootElements, uint32_t * __restrict__ SmallestNonce, uint32_t* result)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint32_t NonceNumber = 1;  // old
	uint32_t ThreadNumber = 32;
	uint32_t event_thread = thread / ThreadNumber;
	uint32_t NonceIterator = startNounce + event_thread;
	//	uint32_t thread_event = thread / event_base; // might be a lot (considering this isn't thread per blocks)
	if (event_thread < threads)
	{



		// pointers
		const uint2 *    __restrict__ BlockHistory = &DBlockHistory[0];
		const uint4 *	 __restrict__ GBlock	   = &DBlock[0];

//if (thread==0)
//	printf("coming here blockhistory %08x %08x \n",BlockHistory[10].x,BlockHistory[10].y);

		__shared__ uint8 YLocal;
		
		if (threadIdx.x%32 == 0) {
		uint16 DataChunk[1] = { 0 };
		DataChunk[0].lo = swapvec(((uint8*)Elements)[0]);
//printf("NonceIterator=%08x event_thread=%08x startNonce=%08x\n",NonceIterator,event_thread,startNounce);

//		DataChunk[0].hi.s0 = (cuda_swab32(NonceIterator) & 0xff000000)  | (0x00800000 & 0xffffff);  // current implementation uint8_t nonce
		DataChunk[0].hi.s0 = (cuda_swab32(NonceIterator));  // uint32_t nonce implementation nonce
		DataChunk[0].hi.s1 = 0x80000000;
		DataChunk[0].sf = 288; // 264; uint8_t nonce
//printf("DataChunk %08x %08x %08x %08x  %08x %08x %08x %08x \n",DataChunk[0].s0, DataChunk[0].s1, DataChunk[0].s2, DataChunk[0].s3, DataChunk[0].s4, DataChunk[0].s5, DataChunk[0].s6, DataChunk[0].s7);
		YLocal = swapvec(sha256_Transform2(DataChunk, H256));
		}
		__syncthreads();

		__shared__ bool init_blocks; 
		__shared__ uint32_t unmatch_block;
		__shared__ uint32_t localIndex;
		init_blocks = false;
		unmatch_block = 0;

		__shared__ uint32_t TestY;
		for (int j = 1; j <= mtp_L; j++)
		{
			if (threadIdx.x%32==0) {
				localIndex = YLocal.s0%(argon_memcost);
			if (j==1) TestY=YLocal.s0;
				if (localIndex == 0 || localIndex == 1) {
					init_blocks = true;
					break;
				}

			}
			__syncthreads();
			__shared__ uint4 X_IJ[64];

			uint2 history = BlockHistory[localIndex];

			uint32_t ref_block  = history.y;
			uint32_t prev_block = history.x;
			fill_block4_doubleshared_mtp_output_bhdr(X_IJ, GBlock, prev_block, ref_block);

			int countIndex;
			for (countIndex = threadIdx.x%32; countIndex < 128; countIndex+=32) {
				if (((uint64_t*)X_IJ)[countIndex] != ((uint64_t*)GBlock)[localIndex * 128 + countIndex]) {
				atomicAdd(&unmatch_block,1);
					break;
				}
			}
			__syncthreads();
			if (unmatch_block>0) break;
			
			if (threadIdx.x%32==0) {

				uint16 DataChunk[1];
				DataChunk[0].lo = swapvec(YLocal);
				DataChunk[0].hi = swapvec(((uint8*)GBlock)[localIndex * 32]);


				uint8 DataTmp = sha256_Transform2(DataChunk, H256);
				for (int i = 0; i < 15; i++) {
					DataChunk[0].lo = swapvec(((uint8*)GBlock)[localIndex * 32 + 2 * i + 1]);
					DataChunk[0].hi = swapvec(((uint8*)GBlock)[localIndex * 32 + 2 * i + 2]);
					DataTmp = sha256_Transform2(DataChunk, DataTmp);
				}
				DataChunk[0].lo = swapvec(((uint8*)GBlock)[localIndex * 32 + 31]);
				DataChunk[0].hi = pad4;
				DataChunk[0].sf = (1024 + 32) * 8;
				YLocal = swapvec(sha256_Transform2(DataChunk, DataTmp));
			}
			__syncthreads();
		}
		// end loop		

		if (init_blocks) {
			return; // not a solution
		}


		if (unmatch_block>0) {
			return; // not a solution
		}
		// search if there are d trailing 0


//		if (((uint64_t*)state)[3] <= ((uint64_t*)pTarget)[3]) {
		if (threadIdx.x % 32 == 0 && (YLocal.s7 <= pTarget[7])) 
		{
		atomicMin(&SmallestNonce[0],NonceIterator);
		}
		__syncthreads();

		if (threadIdx.x%32==0 &&  (YLocal.s7 <= pTarget[7]) && NonceIterator==SmallestNonce[0]) {
			((uint8*)result)[0] = YLocal;
			printf("TestY = %08x\n",TestY);
			printf("Nonce = %08x\n", SmallestNonce[0]);
			printf("thread = %d GPU YLocal final %08x %08x %08x %08x %08x %08x %08x %08x \n",thread/32,YLocal.s0, YLocal.s1, YLocal.s2, YLocal.s3, 
				YLocal.s4, YLocal.s5, YLocal.s6, YLocal.s7);
		


		
		}

	}
}



__host__
void argon2_cpu_init(int thr_id, uint32_t threads)
{

	// just assign the device pointer allocated in main loop

printf("number of threads %d \n",threads);
	hipMalloc((void**)&HBlock, 256 * argon_memcost * sizeof(uint32_t) );
	hipMalloc((void**)&HBlockHistory, argon_memcost * sizeof(uint64_t) );

	hipMalloc(&d_GNonces[thr_id], 8 * sizeof(uint32_t));
	hipHostMalloc(&h_GNonces[thr_id], 8 * sizeof(uint32_t));
	hipMalloc(&d_MinNonces[thr_id], sizeof(uint32_t));
	hipHostMalloc(&h_MinNonces[thr_id],  sizeof(uint32_t));
}


__host__
void argon2_setBlockTarget(const void* pDataIn,const void *pTargetIn, const void * zElement)
{

	printf("the target %08x %08x %08x %08X  %08x %08x %08x %08X \n",
		((uint32_t*)pTargetIn)[0],
		((uint32_t*)pTargetIn)[1],
		((uint32_t*)pTargetIn)[2],
		((uint32_t*)pTargetIn)[3],
		((uint32_t*)pTargetIn)[4],
		((uint32_t*)pTargetIn)[5],
		((uint32_t*)pTargetIn)[6],
		((uint32_t*)pTargetIn)[7]
);
	hipMemcpyToSymbol(HIP_SYMBOL(pData), pDataIn, 32, 0, hipMemcpyHostToDevice); // shortened message
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 32, 0, hipMemcpyHostToDevice);	
	hipMemcpyToSymbol(HIP_SYMBOL(Elements), zElement, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice);

}

__host__
void mtp_fill(const uint64_t *Block, const uint64_t zblockHistory,uint32_t offset)
{
//uint4 TransBlock[64];
//	memcpy(TransBlock,Block,256*sizeof(uint32_t*));
//	hipMemcpyToSymbol(HIP_SYMBOL(HBlock), Block, sizeof(Block), offset*sizeof(Block), hipMemcpyHostToDevice);
uint4 *Blockptr   = &HBlock[offset*64];
uint2 *Historyptr = &HBlockHistory[offset];
	hipMemcpyAsync(Blockptr, Block, 256 * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpyAsync(Historyptr, &zblockHistory, 2 * sizeof(uint32_t), hipMemcpyHostToDevice);
}

__host__
uint32_t argon2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash)
{

	uint32_t result = UINT32_MAX;
	hipMemset(d_GNonces[thr_id], 0xff, 8 * sizeof(uint32_t));
	hipMemset(d_MinNonces[thr_id],0xff,sizeof(uint32_t));
	int dev_id = device_map[thr_id % MAX_GPUS];

	uint32_t tpb = 32; //TPB52;
 
	dim3 gridyloop(threads*32/tpb);
	dim3 blockyloop(tpb);

	mtp_yloop << < gridyloop,blockyloop >> >(threads,startNounce,HBlock,HBlockHistory,Elements,d_MinNonces[thr_id],d_GNonces[thr_id]);

	
	// get first found nonce
	hipMemcpy(h_GNonces[thr_id], d_GNonces[thr_id], 8 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	hipMemcpy(h_MinNonces[thr_id], d_MinNonces[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
if (h_MinNonces[thr_id][0]!=0xffffffff)
printf("Nonce %08x sol one %08x %08x %08x %08x %08x %08x %08x %08x\n",h_MinNonces[thr_id][0], h_GNonces[thr_id][0], 
	  h_GNonces[thr_id][1], h_GNonces[thr_id][2], h_GNonces[thr_id][3], h_GNonces[thr_id][4], h_GNonces[thr_id][5], 
	  h_GNonces[thr_id][6], h_GNonces[thr_id][7]);
else 
printf("startNounce=%08x threads=%08x sum=%08x\n",startNounce,threads,startNounce+threads);
	result = *h_MinNonces[thr_id];
	return result;

}
