#include "hip/hip_runtime.h"


#include "argon2ref/argon2.h"
#include "merkletree/mtp.h"

#include <unistd.h>
#include "miner.h"
#include "cuda_helper.h"
#define memcost 4*1024*1024

extern void mtp_cpu_init(int thr_id, uint32_t threads);

extern uint32_t mtp_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce);

extern void mtp_setBlockTarget(const void* pDataIn, const void *pTargetIn, const void * zElement);
extern void mtp_fill(const uint64_t *Block, uint32_t offset, uint32_t datachunk);

#define HASHLEN 32
#define SALTLEN 16
#define PWD "password"
//#define MTP_L 64

static bool init[MAX_GPUS] = { 0 };
static __thread uint32_t throughput = 0;
static uint32_t JobId = 0;
static MerkleTree::Elements TheElements;
static MerkleTree ordered_tree;
static unsigned char TheMerkleRoot[16];
static argon2_context context;
argon2_instance_t instance;
extern "C" int scanhash_mtp(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done, struct mtp* mtp)
{
//	if (work_restart[thr_id].restart) return 0;
//	unsigned char TheMerkleRoot[16];
	unsigned char mtpHashValue[32];
//	MerkleTree::Elements TheElements; // = new MerkleTree;
printf("the job_id from mtp %s\n",work->job_id+8);
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int dev_id;
	if (opt_benchmark)
		ptarget[7] = 0x00ff;

		uint32_t diff = 5;
		uint32_t TheNonce;

	if (!init[thr_id])
	{
		dev_id = device_map[thr_id];
		hipSetDevice(dev_id);
		
		hipDeviceReset();
//		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
//		hipSetDeviceFlags(hipDeviceScheduleYield);

		int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 17 : 16;
		if (device_sm[device_map[thr_id]] == 500) intensity = 15;
		intensity = 1;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
		throughput =  1024*64;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);


		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);


		mtp_cpu_init(thr_id, throughput);

		init[thr_id] = true;

	}

	uint32_t _ALIGN(128) endiandata[20];
	((uint32_t*)pdata)[19] = (pdata[20]); //*/0x00100000; // mtp version not the actual nonce
//	((uint32_t*)pdata)[19] = 0x1000;

	for (int k = 0; k < 20; k++) 
		endiandata[k] = pdata[k];
	
//	argon2_context context = init_argon2d_param((const char*)endiandata);
//	argon2_instance_t instance;
//	argon2_ctx_from_mtp(&context, &instance);


printf("Pdata1 %08x work->data[1] %08x\n", JobId,work->data[17]);
if (JobId!= work->data[17]){

if (JobId!=0)
	free_memory(&context, (unsigned char *)instance.memory, instance.memory_blocks, sizeof(block));

	context = init_argon2d_param((const char*)endiandata);
	argon2_ctx_from_mtp(&context, &instance);


	TheElements = mtp_init(&instance);
//	MerkleTree ordered_tree;
	ordered_tree = MerkleTree(TheElements, true);
	JobId = work->data[17];

	MerkleTree::Buffer root = ordered_tree.getRoot();
	std::copy(root.begin(), root.end(), TheMerkleRoot);

	mtp_setBlockTarget(endiandata,ptarget,&TheMerkleRoot);
const int datachunk = 512;
	if (work_restart[thr_id].restart) goto TheEnd;

printf("filling memory\n");

for (int i=0;i<(memcost/ datachunk) && !work_restart[thr_id].restart;i++) {
uint64_t *Truc =(uint64_t *) malloc(128* datachunk*sizeof(uint64_t));
	
	for (int j=0;j<datachunk;j++)
		memcpy(&Truc[128*j],instance.memory[datachunk*i+j].v,128*sizeof(uint64_t));

	mtp_fill(Truc, i, datachunk);
	free(Truc);
}
printf("memory filled \n");
}

	if (work_restart[thr_id].restart) goto TheEnd;
		pdata[19] = first_nonce;
do  {
		int order = 0;
		uint32_t foundNonce;

		*hashes_done = pdata[19] - first_nonce + throughput;
	  
		foundNonce = mtp_cpu_hash_32(thr_id, throughput, pdata[19]);

		uint32_t _ALIGN(64) vhash64[8];
		if (foundNonce != UINT32_MAX)
		{

			block_mtpProof TheBlocksAndProofs[140];
			uint256 TheUint256Target[1];
			TheUint256Target[0] = ((uint256*)ptarget)[0];

			blockS nBlockMTP[MTP_L *2];
			unsigned char nProofMTP[MTP_L * 3 * 353 ];
			
			uint32_t is_sol = mtp_solver(foundNonce, &instance, nBlockMTP,nProofMTP, TheMerkleRoot, mtpHashValue, ordered_tree, endiandata,TheUint256Target[0]);

			if (is_sol==1 /*&& fulltest(vhash64, ptarget)*/) {
				int res = 1;
				work_set_target_ratio(work, vhash64);		

				pdata[19] =/*swab32*/(foundNonce);

/// fill mtp structure
				mtp->MTPVersion = 0x1000;
			for (int i=0;i<16;i++) 
				mtp->MerkleRoot[i] = TheMerkleRoot[i];
			for (int i = 0; i<32; i++)
				mtp->mtpHashValue[i] = mtpHashValue[i];
			
			for (int j=0;j<(MTP_L * 2);j++)
				for (int i=0;i<128;i++)
				mtp->nBlockMTP[j][i]= nBlockMTP[j].v[i];
                int lenMax =0; 
				int len = 0;

				memcpy(mtp->nProofMTP, nProofMTP, sizeof(unsigned char)* MTP_L * 3 * 353);


				printf("found a solution, nonce %08x\n",pdata[19]);
//				free_memory(&context, (unsigned char *)instance.memory, instance.memory_blocks, sizeof(block));
				
				return res;

			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}
		work_set_target_ratio(work, vhash64);
/*
		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
*/
		pdata[19] += throughput;
//		be32enc(&endiandata[19], pdata[19]);
	}   while (!work_restart[thr_id].restart && pdata[19]<0xffffffff);

TheEnd:
//	free_memory(&context, (unsigned char *)instance.memory, instance.memory_blocks, sizeof(block));
	*hashes_done = pdata[19] - first_nonce;

//	ordered_tree.~MerkleTree();
//	TheElements.clear();
	return 0;
}


