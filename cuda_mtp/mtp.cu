#include "hip/hip_runtime.h"


#include "argon2ref/argon2.h"
#include "merkletree/mtp.h"

#include <unistd.h>
#include "miner.h"
#include "cuda_helper.h"
#define memcost 4*1024*1024

extern void mtp_cpu_init(int thr_id, uint32_t threads);

extern uint32_t mtp_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce);

extern void mtp_setBlockTarget(int thr_id,const void* pDataIn, const void *pTargetIn, const void * zElement);
extern void mtp_fill(uint32_t d, const uint64_t *Block, uint32_t offset, uint32_t datachunk);

#define HASHLEN 32
#define SALTLEN 16
#define PWD "password"
//#define MTP_L 64

static bool init[MAX_GPUS] = { 0 };
static __thread uint32_t throughput = 0;
static uint32_t JobId[MAX_GPUS] = {0};
//static  MerkleTree::Elements TheElements[MAX_GPUS];
static  MerkleTree ordered_tree[MAX_GPUS];
static  unsigned char TheMerkleRoot[MAX_GPUS][16];
static  argon2_context context[MAX_GPUS];
static argon2_instance_t instance[MAX_GPUS];
//static pthread_mutex_t work_lock;
//static pthread_barrier_t barrier;
extern "C" int scanhash_mtp(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done, struct mtp* mtp)
{
//	if (work_restart[thr_id].restart) return 0;
//	unsigned char TheMerkleRoot[16];
	unsigned char mtpHashValue[32];

//	pthread_mutex_init(&work_lock, NULL);
//	pthread_barrier_init(&barrier, NULL, 1);

//	MerkleTree::Elements TheElements; // = new MerkleTree;
//printf("the job_id from mtp %s\n",work->job_id+8);
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int dev_id = device_map[thr_id];;
	if (opt_benchmark)
		ptarget[7] = 0x00ff;

		uint32_t diff = 5;
		uint32_t TheNonce;

	if (!init[thr_id])
	{

		hipSetDevice(dev_id);
		
		hipDeviceReset();
//		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
//		hipSetDeviceFlags(hipDeviceScheduleYield);

//		int intensity = (device_sm[dev_id] >= 500 && !is_windows()) ? 17 : 16;
//		if (device_sm[device_map[thr_id]] == 500) intensity = 15;
		int intensity = 16;
		throughput = cuda_default_throughput(thr_id, 1U << intensity); // 18=256*256*4;
//		throughput =  1024*64;
		if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, dev_id);


		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);


		mtp_cpu_init(thr_id, throughput);

		init[thr_id] = true;

	}

	uint32_t _ALIGN(128) endiandata[20];
	((uint32_t*)pdata)[19] = (pdata[20]); //*/0x00100000; // mtp version not the actual nonce
//	((uint32_t*)pdata)[19] = 0x1000;

	for (int k = 0; k < 20; k++) 
		endiandata[k] = pdata[k];
	
//	argon2_context context = init_argon2d_param((const char*)endiandata);
//	argon2_instance_t instance;
//	argon2_ctx_from_mtp(&context, &instance);
//printf("coming here\n");


//pthread_mutex_lock(&work_lock);

if (JobId[thr_id]!= work->data[17]){
//restart_threads();
//pthread_barrier_wait(&barrier);
if (JobId[thr_id]!=0)
	free_memory(&context[thr_id], (unsigned char *)instance[thr_id].memory, instance[thr_id].memory_blocks, sizeof(block));

//printf("coming here2\n");
	context[thr_id] = init_argon2d_param((const char*)endiandata);
	argon2_ctx_from_mtp(&context[thr_id], &instance[thr_id]);


	MerkleTree::Elements TheElements = mtp_init2(&instance[thr_id]);

	ordered_tree[thr_id] = MerkleTree(TheElements, true);
	JobId[thr_id] = work->data[17];

	MerkleTree::Buffer root = ordered_tree[thr_id].getRoot();
	std::copy(root.begin(), root.end(), TheMerkleRoot[thr_id]);

//	mtp_setBlockTarget(0,endiandata,ptarget,&TheMerkleRoot);
	mtp_setBlockTarget(thr_id, endiandata, ptarget, &TheMerkleRoot[thr_id]);

printf("filling memory\n");
const int datachunk = 512;
for (int i=0;i<((uint32_t)memcost/ datachunk) /* && !work_restart[thr_id].restart*/;i++) {
uint64_t *Truc =(uint64_t *) malloc(128* datachunk*sizeof(uint64_t));
	
	for (int j=0;j<datachunk;j++)
		memcpy(&Truc[128*j],instance[thr_id].memory[datachunk*i+j].v,128*sizeof(uint64_t));

	mtp_fill(thr_id,Truc, i, datachunk);
//	mtp_fill(1, Truc, i, datachunk);
	free(Truc);
}
printf("memory filled \n");
}
//pthread_mutex_unlock(&work_lock);



	if (work_restart[thr_id].restart) goto TheEnd;
		pdata[19] = first_nonce;
do  {
		int order = 0;
		uint32_t foundNonce;

		*hashes_done = pdata[19] - first_nonce + throughput;
//printf("first nonce %08x thr_id %08x\n", pdata[19],thr_id);

		foundNonce = mtp_cpu_hash_32(thr_id, throughput, pdata[19]);

		uint32_t _ALIGN(64) vhash64[8];
		if (foundNonce != UINT32_MAX)
		{

			block_mtpProof TheBlocksAndProofs[140];
			uint256 TheUint256Target[1];
			TheUint256Target[0] = ((uint256*)ptarget)[0];

			blockS nBlockMTP[MTP_L *2];
			unsigned char nProofMTP[MTP_L * 3 * 353 ];
			
			uint32_t is_sol = mtp_solver(foundNonce, &instance[thr_id], nBlockMTP,nProofMTP, TheMerkleRoot[thr_id], mtpHashValue, ordered_tree[thr_id], endiandata,TheUint256Target[0]);

			if (is_sol==1 /*&& fulltest(vhash64, ptarget)*/) {
				int res = 1;
				work_set_target_ratio(work, vhash64);		

				pdata[19] =/*swab32*/(foundNonce);

/// fill mtp structure
				mtp->MTPVersion = 0x1000;
			for (int i=0;i<16;i++) 
				mtp->MerkleRoot[i] = TheMerkleRoot[thr_id][i];
			for (int i = 0; i<32; i++)
				mtp->mtpHashValue[i] = mtpHashValue[i];
			
			for (int j=0;j<(MTP_L * 2);j++)
				for (int i=0;i<128;i++)
				mtp->nBlockMTP[j][i]= nBlockMTP[j].v[i];
                int lenMax =0; 
				int len = 0;

				memcpy(mtp->nProofMTP, nProofMTP, sizeof(unsigned char)* MTP_L * 3 * 353);


//				printf("found a solution, nonce %08x\n",pdata[19]);
//				free_memory(&context, (unsigned char *)instance.memory, instance.memory_blocks, sizeof(block));
//				pthread_mutex_destroy(&work_lock);
				return res;

			} else {
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}
		work_set_target_ratio(work, vhash64);
/*
		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
*/
		pdata[19] += throughput;
//		be32enc(&endiandata[19], pdata[19]);
	}   while (!work_restart[thr_id].restart && pdata[19]<0xffffffff);

TheEnd:
//	free_memory(&context, (unsigned char *)instance.memory, instance.memory_blocks, sizeof(block));
	*hashes_done = pdata[19] - first_nonce;

//	ordered_tree.~MerkleTree();
//	TheElements.clear();
//	pthread_mutex_destroy(&work_lock);
	return 0;
}


